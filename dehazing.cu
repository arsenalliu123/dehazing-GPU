#include "hip/hip_runtime.h"
#include "dehazing.h"
#include "stdio.h"




//convenient macros
#define IN_GRAPH(x,y,h,w) ((x>=0)&&(x<h)&&(y>=0)&&(y<w))
#define min(x,y) ((x<y)?x:y)
#define max(x,y) ((x>y)?x:y)

/*
 * dark_channel host wrapper and kernel
 */
//first kernel calculate min of RGB
__global__
void dark_channel_kernel(float3 *image, float *dark, int height, int width){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = x * width + y;
	if(x < height && y < width){
		dark[i] = min(image[i].x, min(image[i].y, image[i].z));
	}
}

//second kernel calculate min of 15 X 15 ceil
__global__
void prior_kernel(float *dark, int height, int width, int window){
	extern __shared__ float buffer[];
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = x * width + y;
	if(x < height && y < width){
		const int si = (threadIdx.x + window) * (blockDim.y + window * 2) + threadIdx.y + window;
		buffer[si] = dark[i];
		if(threadIdx.x < window && IN_GRAPH(x-window, y, height, width) ){
			buffer[si - (blockDim.y + window * 2) * window] = dark[i - window * width];
			if(threadIdx.y < window &&
				IN_GRAPH(x-window, y-window, height, width) ){
				buffer[si - (blockDim.y + window * 2) * window - window]
			       = dark[i - window * width - window];
			}
			if(threadIdx.y >= blockDim.y - window &&
				IN_GRAPH(x-window, y+window, height, width) ){
				buffer[si - (blockDim.y + window * 2) * window + window]
			       = dark[i - window * width + window];
			}
		}
		if(threadIdx.x >= blockDim.x - window && IN_GRAPH(x+window, y, height, width) ){
			buffer[si + (blockDim.y + window * 2) * window] = dark[i + window * width];
			if(threadIdx.y >= blockDim.y - window &&
				IN_GRAPH(x+window, y+window, height, width) ){
					buffer[si + (blockDim.y + window * 2) * window + window]
					       = dark[i + window * width + window];
			}
			if(threadIdx.y < window &&
				IN_GRAPH(x+window, y-window, height, width) ){
					buffer[si + (blockDim.y + window * 2) * window - window]
					       = dark[i + window * width - window];
			}

		}
		if(threadIdx.y >= blockDim.y - window && IN_GRAPH(x, y+window, height, width) ){
			buffer[si + window] = dark[i + window];
		}
		if(threadIdx.y < window && IN_GRAPH(x, y-window, height, width) ){
			buffer[si - window] = dark[i - window];
		}

		__syncthreads();
		
		float minval = 1.0;
		for(int startx = 0; startx < window * 2 + 1; startx++){
			for(int starty = 0; starty < window * 2 + 1; starty++){
				if(IN_GRAPH(x-window+startx, y-window+starty, height, width)){
				minval = min(
						buffer[
						       (threadIdx.x+startx)*
						       (blockDim.y + window * 2) +
						       threadIdx.y + starty], minval);
				}
			}
		}
		dark[i] = minval;

	}
}

void dark_channel(float *image,float *dark_channel,int height, int width, dim3 blocks,dim3 grids){
	dark_channel_kernel<<<grids, blocks>>> ((float3 *)image, dark_channel, height, width);
	int window = 7;
	int shared_size = (blocks.x + window * 2) * (blocks.y + window * 2) * sizeof(float);
	prior_kernel<<<grids, blocks, shared_size>>>(dark_channel, height, width, window);
}

/*
 * air_light host wrapper and kernel
 */

//first kernel reduce to < 1024 values for next kernel
__global__
void dehazing_img_kernel1(
		float3 *image, float *dark,
		int height, int width,
		float3 *int_image, float *int_dark){
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	//printf("%d %d %d %d\n", b_n, i, threadIdx.x , width*height);
	extern __shared__ float3 tmp_image[];
	float *tmp_dark = (float *)(tmp_image + blockDim.x);
	if(i < width * height){
		tmp_image[threadIdx.x] = image[i];
		tmp_dark[threadIdx.x] = dark[i];
		__syncthreads();
		for(unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1){
			if(threadIdx.x < stride){
				if(tmp_dark[threadIdx.x + stride] > tmp_dark[threadIdx.x]){
					tmp_dark[threadIdx.x] = tmp_dark[threadIdx.x + stride];
					tmp_image[threadIdx.x] = tmp_image[threadIdx.x + stride];
				}
			}
			__syncthreads();
		}
		if(threadIdx.x == 0){
			int_image[blockIdx.x] = tmp_image[threadIdx.x];
			int_dark[blockIdx.x] = tmp_dark[threadIdx.x];
		}
	}
}

//calculate air light
__global__
void dehazing_img_kernel2(float3 *image, int size, float3 *int_image, float *int_dark){

	extern __shared__ float3 tmp_image[];
	float *tmp_dark = (float *)(tmp_image + blockDim.x);
	tmp_image[threadIdx.x] = int_image[threadIdx.x];
	tmp_dark[threadIdx.x] = int_dark[threadIdx.x];
	__syncthreads();
	for(unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1){
		if(threadIdx.x < stride){
			if(tmp_dark[threadIdx.x + stride] > tmp_dark[threadIdx.x]){
				tmp_dark[threadIdx.x] = tmp_dark[threadIdx.x + stride];
				tmp_image[threadIdx.x] = tmp_image[threadIdx.x + stride];
			}
		}
		__syncthreads();
	}
	if(threadIdx.x == 0){
		image[size] = tmp_image[threadIdx.x];
	}
}

void air_light(float *image, float *dark, int height, int width, dim3 blocks, dim3 grids){
	float3 *int_image = NULL;
	float *int_dark = NULL;
	//printf("%d\n", grids.x);
	hipMalloc((void **)(&int_image), sizeof(float3)*grids.x);
	hipMalloc((void **)(&int_dark), sizeof(float)*grids.x);
	//float *xx = (float *)malloc(sizeof(float)*height*width);
	//CUDA_CHECK_RETURN(hipMemcpy(xx, dark, height * width * sizeof(float), hipMemcpyDeviceToHost));
	//for(int i=0;i<height*width;i++){printf("%.2f ", xx[i]);}
	int shared_size_1 = blocks.x*(sizeof(float3)+sizeof(float));
	int shared_size_2 = grids.x*(sizeof(float3)+sizeof(float));
	dehazing_img_kernel1<<<grids, blocks, shared_size_1>>> ((float3 *)image, dark, height, width, int_image, int_dark);
	dehazing_img_kernel2<<<1, grids, shared_size_2>>> ((float3 *)image, height*width, int_image, int_dark);

}

__global__
void transmission_kernel(float3 *image, float transmission, int height, int width){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = x * width + y;
	if(x < height && y < width){
		tx = image[i].x/image[height*width].x;
		ty = image[i].y/image[height*width].y;
		tz = image[i].z/image[height*width].z;
		transmission[i] = 1 - 0.75*min(tx, min(ty, tz));
	}
}

void transmission(float3 *image, float3 *t, int height, int width, dim3 blocks,dim3 grids){
	transmission_kernel<<<grids, blocks>>> ((float3 *)image, transmission, height, width);
	int window = 7;
	int shared_size = (blocks.x + window * 2) * (blocks.y + window * 2) * sizeof(float);
	prior_kernel<<<grids, blocks, shared_size>>>(transmission, height, width, window);
}

__global__
void dehaze_kernel(float3 *image, float *dark, float t, int height, int width){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = x * width + y;
	if(x < height && y < width){
		image[i].x = (image[i].x - image[height*width].x*(1-t[i])) / t[i];
		image[i].y = (image[i].y - image[height*width].y*(1-t[i])) / t[i];
		image[i].z = (image[i].z - image[height*width].z*(1-t[i])) / t[i];
	}
}

void dehaze(float3 *image,float *dark, float *t, int height, int width, dim3 blocks,dim3 grids){
	dehaze_kernel<<<grids, blocks>>> (image, dark, t, height, width);
}
