#include "hip/hip_runtime.h"
#include "dehazing.h"

using namespace std;
using namespace cv;


/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }



//convinent macros
#define IN_GRAPH(x,y,h,w) ((x>=0)&&(x<h)&&(y>=0)&&(y<w))
#define mymin(x,y) ((x<y)?x:y)
#define mymax(x,y) ((x>y)?x:y)

__global__ void kernel(
		gpu::DevMem2Df mat,
		gpu::DevMem2Df trans_mat,
		float airlight1,
		float airlight2,
		float airlight3,
		gpu::DevMem2Df dest,
		int height, int width, int t0)
{
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	extern __shared__ float airlight[];
	airlight[0] = airlight1;
	airlight[1] = airlight2;
	airlight[2] = airlight3;
	if(x < height && y < width){

		unsigned index_grey = x * width + y;
		unsigned index = (index_grey) * 3;

		for(int i = 0; i < 3; i++){
			dest.data[index+i] =
				(mat.data[index+i] - airlight[i])/mymax(trans_mat.data[index_grey], t0)
				+ airlight[i];
		}
	}

}
void gpu_func(
		gpu::DevMem2Df mat,
		gpu::DevMem2Df trans_mat,
		Vec<float, 3> airlight,
		gpu::DevMem2Df dest,
		int _PriorSize,
		int height,
		int width,
		int t0)
{
	dim3 grid(height/_PriorSize+1,width/_PriorSize+1);
	dim3 block(_PriorSize,_PriorSize);
    kernel<<<grid,block>>>(
    		mat, trans_mat, airlight[0], airlight[1], airlight[2], dest, height, width, t0);
}

//Read Image
/*
void ReadImage(){
	Mat img=imread(img_name, CV_LOAD_IMAGE_COLOR);
	img = img/255;
	height = img.rows;
	width = img.cols;
	size = img.rows*img.cols;
	Mat real_img(img.rows,img.cols,CV_32FC3);
	img.convertTo(real_img,CV_32FC3);
	real_img=real_img/255;


	gpu::GpuMat gpu_img(real_img);
	gpu::GpuMat gpu_channel[3];
	gpu::split(gpu_img, gpu_channel);

	gpu::GpuMat dark_channel(gpu_channel[0]);

	gpu::min(dark_channel, gpu_channel[2], dark_channel);
	gpu::min(dark_channel, gpu_channel[1], dark_channel);
	gpu::erode(dark_channel, dark_channel, Mat::ones(_PriorSize,_PriorSize,0));

	//int n_bright=_topbright*size;
	Point maxLoc;
	gpu::minMaxLoc(dark_channel,0,0,0,&maxLoc);
	Vec<float,3> airlight = real_img.at<Vec<float,3>>(maxLoc);
	gpu::GpuMat gpu_al = gpu::GpuMat(airlight);

	Mat trans_img = imread(trans_name, 0);
	Mat real_trans_img(img.rows,img.cols,CV_32FC1);
	trans_img.convertTo(real_trans_img,CV_32FC1);
	real_trans_img=real_trans_img/255;

	gpu::GpuMat gpu_trans_img(real_trans_img);
	gpu::GpuMat gpu_dest(height, width, CV_32FC3);
	gpu_func(gpu_img, gpu_trans_img, gpu_al, gpu_dest);

	Mat dest(height, width, CV_32FC3);
	gpu_dest.download(dest);

	imwrite(out_name,free_img*255);
	Image *image = (Image *)malloc(size*sizeof(Image));

	for(int i=0;i<height;i++){
		for(int j=0;j<width;j++){
			image[i*width+j].red = img.at<float>(i,j,0);
			image[i*width+j].blue = img.at<float>(i,j,1);
			image[i*width+j].green = img.at<float>(i,j,2);
		}
	}

	return image;
}
*/
//Read TransImage
/*
TransImage* ReadTransImage(){
	Mat img=imread(img_name, 0);
	Mat real_img(img.rows,img.cols,CV_32FC1);
	img.convertTo(real_img,CV_32FC1);
	TransImage *image = (TransImage *)malloc(size*sizeof(TransImage));
	for(int i=0;i<height;i++){
		for(int j=0;j<width;j++){
			image[i*width+j].grey = real_img.at<float>(i,j);
		}
	}

	return image;
}
*/
//Calculate Dark Channel
//J^{dark}(x)=min( min( J^c(y) ) )
/*
__global__
void DarkChannel(Image *image, int height, int width, int erosion_width)
{

	GpuMat dark=GpuMat::zeros(img.rows,img.cols,CV_32FC1);
	GpuMat dark_out=GpuMat::zeros(img.rows,img.cols,CV_32FC1);
	for(int i=0;i<img.rows;i++)
	{
		for(int j=0;j<img.cols;j++)
		{
			dark.at<float>(i,j)=min(min(img.at<Vec<float,3>>(i,j)[0],img.at<Vec<float,3>>(i,j)[1]),min(img.at<Vec<float,3>>(i,j)[0],img.at<Vec<float,3>>(i,j)[2]));
		}
	}
	erode(dark,dark_out,Mat::ones(_PriorSize,_PriorSize,CV_32FC1));
	return dark_out;


	__shared__ float buffer[];

	int startx = blockIdx.x * blockDim.x;
	int starty = blockIdx.y * blockDim.y;

	int x = startx + threadIdx.x;
	int y = starty + threadIdx.y;


	int tid = x*height+y;
	int above_tid = tid-7*width;
	if(threadIdx.x < 7){
		if(IN_GRAPH(x-7,y,height,width)){
			buffer[threadIdx.x*(blockDim.y+14)+(threadIdx.y+7)] =
					min(min(image[above_tid].green, image[above_tid].red),image[above_id].blue);
		}
		else{
			buffer[threadIdx.x*(blockDim.y+14)+(threadIdx.y+7)] = 0;
		}
	}
	if(threadIdx.x > blockDim.x - 8){

	}

	if(IN_GRAPH(x,y,height,width)){
		buffer[(threadIdx.x+7)*(blockDim.y+14)+(threadIdx.y+7)] =
				min(min(image[tid].green, image[tid].red),image[tid].blue);
	}


}
*/


//Calculate Airlight
/*
Vec<float,3> Airlight(Mat img, Mat dark)
{
	int n_bright=_topbright*size;
	Mat dark_1=dark.reshape(1,size);
	Vector<int> max_idx;
	float max_num=0;
	int max_pos=0;
	Vec<float,3> a;
	Vec<float,3> A(0,0,0);
	Mat RGBPixcels=Mat::ones(n_bright,1,CV_32FC3);
	Mat HLSPixcels=Mat::ones(n_bright,1,CV_32FC3);
	Mat IdxPixcels=Mat::ones(n_bright,1,CV_32SC1);


	for(int i=0;i<n_bright;i++)
	{
		max_num=0;
		max_idx.push_back(max_num);
		for(float * p = (float *)dark_1.datastart;p!=(float *)dark_1.dataend;p++)
		{
			if(*p>max_num)
			{
				max_num = *p;
				max_idx[i] = (p-(float *)dark_1.datastart);
				RGBPixcels.at<Vec<float,3>>(i,0) = ((Vec<float,3> *)img.data)[max_idx[i]];
				IdxPixcels.at<int>(i,0) = (p-(float *)dark_1.datastart);
				//((Vec<float,3> *)img.data)[max_idx[i]] = Vec<float,3>(0,0,1);
			}
		}
		((float *)dark_1.data)[max_idx[i]]=0;
	}

	float maxL=0.0;
	//int maxIdx=0;
	for(int j=0; j<n_bright; j++)
	{
		A[0]+=RGBPixcels.at<Vec<float,3>>(j,0)[0];
		A[1]+=RGBPixcels.at<Vec<float,3>>(j,0)[1];
		A[2]+=RGBPixcels.at<Vec<float,3>>(j,0)[2];
	}

	A[0]/=n_bright;
	A[1]/=n_bright;
	A[2]/=n_bright;

	return A;
}
*/

//Calculate Transmission Matrix
/*
Mat TransmissionMat(Mat dark)
{

	return 1-_w*dark;
}
*/

//Calculate Haze Free Image
/*
Mat hazefree(Mat img,Mat t,Vec<float,3> a,float exposure = 0)
{
	Mat freeimg=Mat::zeros(height,width,CV_32FC3);
	img.copyTo(freeimg);
	Vec<float,3> * p=(Vec<float,3> *)freeimg.datastart;
	float * q=(float *)t.datastart;
	for(;p<(Vec<float,3> *)freeimg.dataend && q<(float *)t.dataend;p++,q++)
	{
		(*p)[0]=((*p)[0]-a[0])/std::max(*q,t0)+a[0] + exposure;
		(*p)[1]=((*p)[1]-a[1])/std::max(*q,t0)+a[1] + exposure;
		(*p)[2]=((*p)[2]-a[2])/std::max(*q,t0)+a[2] + exposure;
	}
	return freeimg;
}
*/

//************* Utility Functions **********
//Print Matrix

	/*
	cout<<"Reading Image ..."<<endl;
	start=clock();

	//Read image
	Image *image = ReadImage();
	Image *device_image;
	hipMalloc(&device_image, size*sizeof(Image));
	hipMemcpy(device_image, image, size*sizeof(Image), hipMemcpyHostToDevice);

	//Read Trans image
	TransImage *t_image = ReadImage();
	Image *device_t_image;
	hipMalloc(&device_t_image, size*sizeof(TransImage));
	hipMemcpy(device_t_image, t_image, size*sizeof(TransImage), hipMemcpyHostToDevice);

	//finished
	finish=clock();
	duration1=( double )( finish - start )/ CLOCKS_PER_SEC ;
	cout<<"Time Cost: "<<duration1<<"s"<<endl;
	waitKey(1000);
	cout<<endl;

	//Calculate DarkChannelPrior
	cout<<"Calculating Dark Channel Prior ..."<<endl;
	start=clock();
	dark_channel=DarkChannelPrior(img);
	//imshow("Dark Channel Prior",dark_channel);
	//printMatInfo("dark_channel",dark_channel);
	finish=clock();
	duration3=( double )( finish - start )/ CLOCKS_PER_SEC ;
	cout<<"Time Cost: "<<duration3<<"s"<<endl;
	waitKey(1000);
	cout<<endl;

	//Calculate Airlight
	cout<<"Calculating Airlight ..."<<endl;
		start=clock();
	Vec<float,3> a=Airlight(img,dark_channel);
	cout<<"Airlight:\t"<<" B:"<<a[0]<<" G:"<<a[1]<<" R:"<<a[2]<<endl;
		finish=clock();
		duration4=( double )( finish - start )/ CLOCKS_PER_SEC ;
		cout<<"Time Cost: "<<duration4<<"s"<<endl;
	cout<<endl;

	//Reading Refine Trans
	cout<<"Reading Refine Transmission..."<<endl;
	trans_refine=ReadTransImage();
	printMatInfo("trans_refine",trans_refine);
	//imshow("Refined Transmission Mat",trans_refine);
	cout<<endl;

	//Haze Free
	cout<<"Calculating Haze Free Image ..."<<endl;
		start=clock();

	free_img=hazefree(img,trans_refine,a,0.2);
	//imshow("Haze Free",free_img);

	printMatInfo("free_img",free_img);
		finish=clock();
		duration7=( double )( finish - start )/ CLOCKS_PER_SEC ;
		cout<<"Time Cost: "<<duration7<<"s"<<endl;

		//cout<<"Total Time Cost: "<<duration1+duration2+duration3+duration4+duration5+duration6+duration7<<"s"<<endl;

	//Save Image
	//char img_name_dark[100]="Dark_";
	//char img_name_step[100]="Step_";
	//char img_name_free[100]="Hazefree_";
	//strcat(img_name_free,img_name);
	//strcat(img_name_step,img_name);
	//strcat(img_name_dark,img_name);
	imwrite(out_name,free_img*255);
	//imwrite(img_name_step,trans_refine*255);
	//imwrite(img_name_dark,trans*255);
	cout<<"Image saved as "<<out_name<<endl;
	//waitKey();
	cout<<endl;

	return 0;
	*/
